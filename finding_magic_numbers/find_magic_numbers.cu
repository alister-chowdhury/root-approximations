#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <tuple>
#include <typeinfo>

#ifndef TARGET_ROOT
#define TARGET_ROOT 2
#endif

#ifndef FIND_DOUBLE

#define TARGET_FTYPE float
#define TARGET_ITYPE uint32_t
#define MANTISSA_SIZE ((1 << 24) - 1)
#define INITIAL_MAGIC (((127 - 127 / TARGET_ROOT) - 1) << 23)
#define SKIP_AMOUNT_MUL 1

#else

#define TARGET_FTYPE double
#define TARGET_ITYPE uint64_t
#define MANTISSA_SIZE ((IntT(1) << 53) - 1)
#define INITIAL_MAGIC ((IntT(1023 - 1023 / TARGET_ROOT) - 1) << 52)
#define SKIP_AMOUNT_MUL 179424673

#endif

template <uint32_t n, typename T>
__device__ inline T ct_pow(const T x)
{
    if (n == 0) {
        return 1;
    } else {
        const T part = ct_pow<n / 2, T>(x);
        return ((n & 1) ? (x * part * part) : (part * part));
    }
}

template <uint32_t n, typename T>
__device__ T halley_step(const T x0, const T value)
{
    const T fx = ct_pow<n>(x0) - value;
    const T fpx = n * ct_pow<n - 1>(x0);
    const T fppx = n * (n - 1) * ct_pow<n - 2>(x0);
    const T numer = 2 * fx * fpx;
    const T denom = 2 * fpx * fpx - fx * fppx;
    const T x1 = x0 - (numer / denom);
    return x1;
}

template <uint32_t n, typename T>
__device__ T newton_step(const T x0, const T value)
{
    // x1 = x0 - (f(x0)-y)/f'(x0)
    const T x1 = x0 - ((ct_pow<n>(x0) - value) / (n * ct_pow<n - 1>(x0)));
    return x1;
}

template <typename FloatT, typename IntT>
struct SetInitialErrorLevel {
    __device__ void operator()(const IntT index)
    {
        magic_max_error[index] = 10000;
    }
    FloatT* __restrict__ magic_max_error;
};

template <typename FloatT, typename IntT, uint32_t n>
struct FindErrorForMagics {
    __device__ void operator()(const IntT index)
    {
        IntT magic = INITIAL_MAGIC + magic_offset + index;

        FloatT error = 0.0;

        union {
            FloatT f;
            IntT i;

        } packed;

        for (FloatT root_value = 0.001; root_value < 4.0001; root_value += 0.001) {
            const FloatT powered = ct_pow<n>(root_value);

            packed.f = powered;
            packed.i = magic + (packed.i / n);

            const FloatT approx = halley_step<n>(packed.f, powered);

            const FloatT current_error = abs((approx - root_value) / root_value);
            error = max(current_error, error);
        }

        // Reuse memory
        if (error < magic_max_error[index]) {
            magic_max_error[index] = error;
            magics[index] = magic;
        }
    }

    const IntT magic_offset;
    FloatT* __restrict__ magic_max_error;
    IntT* __restrict__ magics;
};

std::pair<TARGET_FTYPE, TARGET_ITYPE> find_magic_number(void)
{
    using FloatT = TARGET_FTYPE;
    using IntT = TARGET_ITYPE;

    using KernelT = FindErrorForMagics<FloatT, IntT, TARGET_ROOT>;

    const IntT max_per_round = (1 << 14);

    // Allocate memory
    FloatT* local_error = new FloatT[max_per_round];
    IntT* local_magics = new IntT[max_per_round];

    FloatT* gpu_errors;
    IntT* gpu_magics;

    hipMalloc(&gpu_errors, sizeof(FloatT) * max_per_round);
    hipMalloc(&gpu_magics, sizeof(IntT) * max_per_round);

    // Set the initial error level to be high
    thrust::for_each(thrust::counting_iterator<IntT>(0),
        thrust::counting_iterator<IntT>(max_per_round),
        SetInitialErrorLevel<FloatT, IntT>{ gpu_errors });

    // Find the best magic
    for (IntT magic_offset = 0; magic_offset < MANTISSA_SIZE;
         magic_offset += (max_per_round * SKIP_AMOUNT_MUL)) {
        thrust::for_each(thrust::counting_iterator<IntT>(0),
            thrust::counting_iterator<IntT>(max_per_round),
            KernelT{ magic_offset, gpu_errors, gpu_magics }

            );
        // Print progress
        std::cerr << std::fixed << ((100.0 * magic_offset) / MANTISSA_SIZE)
                  << "%                    \r";
    }
    std::cerr << "\n";

    // Copy stuff back
    hipMemcpy(local_error, gpu_errors, sizeof(FloatT) * max_per_round,
        hipMemcpyDeviceToHost);
    hipMemcpy(local_magics, gpu_magics, sizeof(FloatT) * max_per_round,
        hipMemcpyDeviceToHost);

    // Find the best magic
    FloatT max_error = 1000;
    IntT current_magic = 0;

    for (IntT i = 0; i < max_per_round; ++i) {
        if (local_error[i] < max_error) {
            max_error = local_error[i];
            current_magic = local_magics[i];
        }
    }

    hipFree(&gpu_errors);
    hipFree(&gpu_magics);
    delete[] local_error;
    delete[] local_magics;

    return std::make_pair(max_error, current_magic);
}

int main(void)
{
    std::cout << "Finding x^" << TARGET_ROOT << " for type ";
#ifdef FIND_DOUBLE
    std::cout << "Double\n";
#else
    std::cout << "Float\n";
#endif
    auto result = find_magic_number();
    std::cout << "Relative error: " << std::fixed << result.first << "\n";
    std::cout << "Magic:          0x" << std::hex << result.second << " + (i / "
              << std::dec << TARGET_ROOT << ")\n";
}